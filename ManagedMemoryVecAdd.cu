#include "hip/hip_runtime.h"
// sudo nvprof --unified-memory-profiling off ./ManagedMemoryVecAdd
// Use this command for profiling without errors for unified memory profiling

#include<iostream>

__global__ void vecAdd(int *a, int *b, int *c, int N){
int i = blockDim.x * blockIdx.x + threadIdx.x;

if(i < N){
c[i] = a[i] + b[i];
}
}

__global__ void squareVec(int *a, int *b, int N){
int i = blockDim.x * blockIdx.x + threadIdx.x;

if(i < N){
b[i] = a[i]*a[i];
}
}


int main(){

int N = 20;
size_t size = N * sizeof(int);

int *a, *b, *c;
hipMallocManaged(&a, size);							// Unified memory; ALWAYS use cudaMemPrefetchAync() with Unified memory to reduce overhead time
hipMallocManaged(&b, size);
hipMallocManaged(&c, size);

for(auto i = 0; i < N; i ++){
a[i] = i;
b[i] = 2*i;
}

int id = hipGetDevice(&id);							// Get the device ID
hipMemPrefetchAsync(a, size, id);						// Use the device ID to prefetch 'a' to the GPU memory
hipMemPrefetchAsync(b, size, id);
hipMemPrefetchAsync(c, size, id);

int NumThreadsPerBlock = 256;
int BlockSize = (N + NumThreadsPerBlock -1)/NumThreadsPerBlock;
vecAdd<<<BlockSize, NumThreadsPerBlock>>>(a, b, c, N);
hipDeviceSynchronize();							// Sunchronize all the threads before moving forward

hipMemPrefetchAsync(a, size, hipCpuDeviceId);				// Prefetch 'a' to the CPU memory; directly use built-in function hipCpuDeviceId
hipMemPrefetchAsync(b, size, hipCpuDeviceId);
hipMemPrefetchAsync(c, size, hipCpuDeviceId);

std::cout << "Printing the vector" << std::endl;
for(auto i = 0; i < N; i++){
std::cout << c[i] << std::endl;
}


hipFree(a);
hipFree(b);

int *c_squared;
hipMallocManaged(&c_squared, size);

int id2 = hipGetDevice(&id);
hipMemPrefetchAsync(c, size, id);
hipMemPrefetchAsync(c_squared, size, id2);

squareVec<<<BlockSize, NumThreadsPerBlock>>>(c,c_squared, N);
hipDeviceSynchronize();

hipMemPrefetchAsync(c, size, hipCpuDeviceId);
hipMemPrefetchAsync(c_squared, size, hipCpuDeviceId);

std::cout << "Printing the vector squared" << std::endl;
for(auto i = 0; i < N; i++){
std::cout << c_squared[i] << std::endl;
}

hipFree(c_squared);
hipFree(c);

return 0;
}
