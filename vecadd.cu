#include<iostream>
#include<hip/hip_runtime.h>

// Device code

__global__ void VecAdd(float* A, float* B, float* C, int N){
    	int i = blockDim.x * blockIdx.x + threadIdx.x;
    
	    if(i < N){
	    	C[i] = A[i] + B[i];
    	}
}

// Host code

int main(){
	int N = 10;
	size_t size = N*sizeof(float);
	
	// Allocate memory for the host
	// float* h_A = (float*)malloc(size);
	// float* h_B = (float*)malloc(size);
	// float* h_C = (float*)malloc(size);
	
	
	// Another way of writing - Pinned memory
	float *h_A, *h_B, *h_C;
	hipHostMalloc(&h_A, size, hipHostMallocDefault);
	hipHostMalloc(&h_B, size, hipHostMallocDefault);
	hipHostMalloc(&h_C, size, hipHostMallocDefault);
	
	// Initialize the input vectors
	for(auto i = 0; i < N; i++){
		h_A[i] = i;
		h_B[i] = 2*i;
		h_C[i] = 0;
	}
	
	// Allocate memory for the device
	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);
	
	// Copy contents of host to device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
	
	// Invoke the kernel to do the computation
	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	
	// Copy results from device to host
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	
	// Free the memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	// Print values
	for(auto i = 0; i < N; i++){
		std::cout << h_C[i] << std::endl;
	}
	
	// Free Pinned memory
	hipFree(h_A);
	hipFree(h_B);
	hipFree(h_C);

	return 0;
}
