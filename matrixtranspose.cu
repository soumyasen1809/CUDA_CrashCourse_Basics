
#include <hip/hip_runtime.h>
#include<iostream>

__global__ void transKernel(int *A, int *A_t, int N){
	int x_index = threadIdx.x + blockIdx.x*blockDim.x;
	int y_index = threadIdx.y + blockIdx.y*blockDim.y;
	
	if(x_index < N && y_index < N){
		A_t[x_index*N+y_index] = A[y_index*N+x_index];
	}
}

int main(){
	int N = 256;
	int size = N*N*sizeof(int);
	
	int A[N][N], At[N][N];
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			A[i][j] = rand()%10;
		}
	}
	
	int *d_A, *d_At;
	
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_At, size);
	
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock (8,8);
	dim3 blocksPerGrid (N/8, N/8);
	transKernel<<<threadsPerBlock, blocksPerGrid>>>(d_A, d_At, N);
	
	hipMemcpy(At, d_At, size, hipMemcpyDeviceToHost);
	
	// for(int i = 0; i < N; i++){
	// 	for(int j = 0; j < N; j++){
	// 		std::cout << At[i][j] << std::endl;
	// 	}
	// }
	
	hipFree(d_A);
	hipFree(d_At);

	return 0;
}
