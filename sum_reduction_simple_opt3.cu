// sudo nvprof --print-gpu-trace --log-file test.txt ./sum_reduction_simple_opt3
// Prints log in txt file


#include <hip/hip_runtime.h>
#include<iostream>
#include<vector>

const int sharedMem = 256*sizeof(double);

__global__ void redSum(double *a, double *out){

	__shared__ double red_mat[sharedMem];
	auto i = (blockDim.x*2)*blockIdx.x + threadIdx.x;
	
	red_mat[threadIdx.x] = a[i]+a[i+blockDim.x];
	__syncthreads();
	
	for(auto k = blockDim.x/2; k > 0; k/=2){
		if(threadIdx.x < k){
			red_mat[threadIdx.x] += red_mat[threadIdx.x+k];
		}
	}
	__syncthreads();
	
	if(threadIdx.x == 0){
		out[blockIdx.x] = red_mat[threadIdx.x];
	}
}


int main(){

int N = 32768;
size_t size = N *sizeof(double);

std::vector<double> h_a(N);
std::vector<double> h_out(N, 0.0);

for(auto i = 0; i < N; i++){
	h_a[i] = 2*i;
}

double *d_a, *d_out;
hipMalloc(&d_a, size);
hipMalloc(&d_out, size);

hipMemcpy(d_a, h_a.data(), size, hipMemcpyHostToDevice);

int threadsPerBlock = 256;		// Launching half the number of threads
int blocksPerGrid = N/threadsPerBlock/2;
redSum<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_out);

hipMemcpy(h_out.data(), d_out, size, hipMemcpyDeviceToHost);

std::cout << h_out[0] << std::endl;

hipFree(d_a);
hipFree(d_out);

return 0;
}
