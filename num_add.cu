// Add 2 numbers


#include <hip/hip_runtime.h>
#include<iostream>

__global__ void numadd(float *d_a, float *d_b, float *d_c){
// int i = blockDim.x * blockIdx.x + threadIdx.x;
*d_c = *d_a + *d_b;

}

int main(){

float h_a, h_b, h_c;
std::cout << "Enter a number" << std::endl;
std::cin >> h_a;
std::cout << "Enter another number" << std::endl;
std::cin >> h_b;

size_t size = sizeof(float);
float *d_a, *d_b, *d_c;
hipMalloc(&d_a, size);
hipMalloc(&d_b, size);
hipMalloc(&d_c, size);

hipMemcpy(d_a, &h_a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, &h_b, size, hipMemcpyHostToDevice);

int numBlocks = 1;
int threadsPerBlock = 1;
numadd<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);

hipMemcpy(&h_c, d_c, size, hipMemcpyDeviceToHost);

std::cout << "The sum is: "<< h_c << std::endl;


hipFree(d_a);
hipFree(d_b);
hipFree(d_c);

return 0;
}
